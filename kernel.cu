#include "hip/hip_runtime.h"
#include "kernel.h"

// Globální proměnné
__constant__ __device__ digit_t d_invN;
__device__ biguint_t d_N;
__device__ biguint_t d_3N;


__global__ void edwardsAdd(ExtendedPoint* R, ExtendedPoint *P, ExtendedPoint *Q)
{
	// Proměnné ve sdílené paměti pro bod P
    __shared__ VOL digit_t x1[NUM_CURVES][NB_DIGITS];
	__shared__ VOL digit_t y1[NUM_CURVES][NB_DIGITS];
	__shared__ VOL digit_t z1[NUM_CURVES][NB_DIGITS];
	__shared__ VOL digit_t t1[NUM_CURVES][NB_DIGITS];
	
	// Proměnné ve sdílené paměti pro bod Q
	__shared__ VOL digit_t x2[NUM_CURVES][NB_DIGITS];
	__shared__ VOL digit_t y2[NUM_CURVES][NB_DIGITS];
	__shared__ VOL digit_t z2[NUM_CURVES][NB_DIGITS];
	__shared__ VOL digit_t t2[NUM_CURVES][NB_DIGITS];
	
	// Pomocné proměnné ve sdílené paměti pro přenos a t0,t1,t2
	__shared__ VOL carry_t carry[NUM_CURVES][NB_DIGITS]; 
	__shared__ VOL digit_t temp0[NUM_CURVES][NB_DIGITS];
	__shared__ VOL digit_t temp1[NUM_CURVES][NB_DIGITS];
	__shared__ VOL digit_t temp2[NUM_CURVES][NB_DIGITS];
	
	VOL digit_t* c_x1 = x1[threadIdx.y];
	VOL digit_t* c_y1 = y1[threadIdx.y];
	VOL digit_t* c_z1 = z1[threadIdx.y];
	VOL digit_t* c_t1 = t1[threadIdx.y];
	
	VOL digit_t* c_x2 = x2[threadIdx.y];
	VOL digit_t* c_y2 = y2[threadIdx.y];
	VOL digit_t* c_z2 = z2[threadIdx.y];
	VOL digit_t* c_t2 = t2[threadIdx.y];
	
	// Pomocné proměnné a konstanty
	VOL digit_t* c_tt0  = temp0[threadIdx.y];   // t0
	VOL digit_t* c_tt1  = temp1[threadIdx.y];   // t1
	
	VOL digit_t* _CARRY = carry[threadIdx.y];  // přenos
	VOL digit_t* _AUX   = temp2[threadIdx.y];  // pomocná proměnná pro násobení
	
	const digit_t _N    = d_N[threadIdx.x];	   // x-tá cifra N
	const digit_t _3N   = d_3N[threadIdx.x];   // x-tá cifra 3*N
	const digit_t _INVN = d_invN;			   // -N^(-1) mod W
	
	// Nakopírování pracovních dat
	const digit_t idx = blockIdx.x*blockDim.y + threadIdx.y;
	
	c_x1[threadIdx.x] = P[idx].C.X[threadIdx.x];
	c_y1[threadIdx.x] = P[idx].C.Y[threadIdx.x];
	c_z1[threadIdx.x] = P[idx].C.Z[threadIdx.x];
	c_t1[threadIdx.x] = P[idx].C.T[threadIdx.x];

	c_x2[threadIdx.x] = Q[idx].C.X[threadIdx.x];
	c_y2[threadIdx.x] = Q[idx].C.Y[threadIdx.x];
	c_z2[threadIdx.x] = Q[idx].C.Z[threadIdx.x];
	c_t2[threadIdx.x] = Q[idx].C.T[threadIdx.x];

	c_cy[threadIdx.x] = 0;
	c_t0[threadIdx.x] = P[idx].y[threadIdx.x]; // t0 = Y1
	c_t1[threadIdx.x] = 0; 
	c_t2[threadIdx.x] = 0; 

	// Twisted Edwards Extended (add-2008-hwcd-4), a = -1, independent of d,incomplete
	/////////////////////////////////////////	
	
	SUE_MOD(c_tt0,c_x1);
	ADD_MOD(c_tt1,c_y2,c_x2);
	
	MUL_MOD(c_tt0,c_tt0,c_tt1);
	ADD_MOD(c_tt1,c_y1,c_x1);
	
	SUB_MOD(c_x1,c_y2,c_x2);
	MUL_MOD(c_tt1,c_tt1,c_x1);
	
	DBL_MOD(c_z2);
	DBL_MOD(c_t2);
	
	MUL_MOD(c_z1,c_z1,c_t2);
	MUL_MOD(c_z2,c_z2,c_t1);
	
	ADD_MOD(c_y2,c_z2,c_z1);
	SUB_MOD(c_x2,c_z2,c_z1);
	
	SUB_MOD(c_z2,c_tt1,c_tt0);
	ADD_MOD(c_t2,c_tt1,c_tt2);
	
	MUL_MOD(c_x1,c_y2,c_z2);
	MUL_MOD(c_y1,c_t2,c_x2);
	MUL_MOD(c_t1,c_y2,c_x2);
	MUL_MOD(c_z1,c_z2,c_t2);
	
	/////////////////////////////////////////
	R[idx].C.X[threadIdx.x] = c_x1[threadIdx.x];
	R[idx].C.Y[threadIdx.x] = c_y1[threadIdx.x];
	R[idx].C.Z[threadIdx.x] = c_z1[threadIdx.x];
	R[idx].C.T[threadIdx.x] = c_t1[threadIdx.x];
}

__global__ void edwardsDbl(ExtendedPoint*R, ExtendedPoint *P)
{
    // Proměnné ve sdílené paměti pro bod P
    __shared__ VOL digit_t x1[NUM_CURVES][NB_DIGITS];
	__shared__ VOL digit_t y1[NUM_CURVES][NB_DIGITS];
	__shared__ VOL digit_t z1[NUM_CURVES][NB_DIGITS];
	__shared__ VOL digit_t t1[NUM_CURVES][NB_DIGITS];
	
	// Pomocné proměnné ve sdílené paměti pro přenos a t0,t1,t2
	__shared__ VOL carry_t carry[NUM_CURVES][NB_DIGITS]; 
	__shared__ VOL digit_t temp0[NUM_CURVES][NB_DIGITS];
	__shared__ VOL digit_t temp1[NUM_CURVES][NB_DIGITS];
	__shared__ VOL digit_t temp2[NUM_CURVES][NB_DIGITS];
	
	VOL digit_t* c_x1 = x1[threadIdx.y];
	VOL digit_t* c_y1 = y1[threadIdx.y];
	VOL digit_t* c_z1 = z1[threadIdx.y];
	VOL digit_t* c_t1 = t1[threadIdx.y];
		
	// Pomocné proměnné a konstanty
	VOL digit_t* c_tt0  = temp0[threadIdx.y];   // t0
	VOL digit_t* c_tt1  = temp1[threadIdx.y];   // t1
	
	VOL digit_t* _CARRY = carry[threadIdx.y];  // přenos
	VOL digit_t* _AUX   = temp2[threadIdx.y];  // pomocná proměnná pro násobení
	
	const digit_t _N    = d_N[threadIdx.x];	   // x-tá cifra N
	const digit_t _3N   = d_3N[threadIdx.x];   // x-tá cifra 3*N
	const digit_t _INVN = d_invN;			   // -N^(-1) mod W
	
	// Nakopírování pracovních dat	
	c_x1[threadIdx.x] = P[idx].C.X[threadIdx.x];
	c_y1[threadIdx.x] = P[idx].C.Y[threadIdx.x];
	c_z1[threadIdx.x] = P[idx].C.Z[threadIdx.x];
	c_t1[threadIdx.x] = P[idx].C.T[threadIdx.x];

	c_cy[threadIdx.x] = 0;
	c_t0[threadIdx.x] = 0;
	c_t1[threadIdx.x] = 0; 
	c_t2[threadIdx.x] = 0; 
 
	// Twisted Edwards Extended (dbl-2008-hwcd-4), a = -1, independent of d,incomplete
	/////////////////////////////////////////
	
	ADD_MOD(c_tt0,c_x1,c_y1);
	SQR_MOD(c_tt1,c_x1);

	SQR_MOD(c_x1,c_y1);
	SQR_MOD(c_y1,c_z1);

	ADD_MOD(c_t1,c_tt1,c_x1);
	SUB_MOD(c_z1,c_tt1,c_x1);

	SQR_MOD(c_tt1,c_tt0);
	DBL_MOD(c_y1);

	SUB_MOD(c_tt0,c_t1,c_tt1);
	ADD_MOD(c_tt1,c_y1,c_z1);

	MUL_MOD(c_x1,c_tt1,c_tt0);
	MUL_MOD(c_y1,c_t1,c_z1);
	MUL_MOD(c_t1,c_t1,c_tt0);
	MUL_MOD(c_z1,c_z1,c_tt1);
	
	////////////////////////////////////////
	R[idx].C.X[threadIdx.x] = c_x1[threadIdx.x];
	R[idx].C.Y[threadIdx.x] = c_y1[threadIdx.x];
	R[idx].C.Z[threadIdx.x] = c_z1[threadIdx.x];
	R[idx].C.T[threadIdx.x] = c_t1[threadIdx.x];
}

void aux_getPointMultiples(ExtendedPoint* R,ExtendedPoint *P,const unsigned int multiple)
{
	edwardsDbl<<NUM_CURVES,NB_DIGITS>>(R,P);
	if (multiple == 2) return;
	for (int i = 3;i <= multiple;++i){
	  edwardsAdd<<NUM_CURVES,NB_DIGITS>>(R,R,P);
	}
}

int buildFromNAF(NAF N,int start,int end)
{
	int i,ret = 0;
	for (i = start;i <= end;i++)
	{
		ret += N.bits[i]*(1 << (i-start));
	}

	return ret;
}

void getPrecomputed(const ExtendedPoint** prec,const int exp,ExtendedPoint** pR)
{
	int k = ((exp > 0 ? exp : -exp)-1)/2;
	if (exp > 0){
	 *pR = prec[k];
	//else {
		//mpz_neg(res,precomp[k]);
	//}
}

extern "C" hipError_t computeExtended(const h_Aux input,h_ExtendedPoint* initPoints,const NAF coeff)
{
	hipError_t cudaStatus;
    
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return cudaStatus;
    }

	// Konstanty do konstantní paměti
	hipMemcpyToSymbol(HIP_SYMBOL(d_N),   (void*)input.N, MAX_BYTES);
	hipMemcpyToSymbol(HIP_SYMBOL(d_3N),  (void*)input.N3,MAX_BYTES);
	hipMemcpyToSymbol(HIP_SYMBOL(d_invN),(void*)input.invN, SIZE_DIGIT/8);

	// Nakopírovat výchozí body do paměti GPU
	ExtendedPoint **pts = new ExtendedPoint[NUM_CURVES];
	for (int i = 0;i < NUM_CURVES;++i){
	   pts[i] = new ExtendedPoint();
	   pts[i].toGPU(initPoints[i]);
	}
    
    // Předpočítat body pro sliding window
    int precompSize = (1 << (coeff.w-2))+1;
    ExtendedPoint **prec = new ExtendedPoint[precompSize*NUM_CURVES];
    for (int i = 0; i < precompSize;++i){
	   prec[i] = new ExtendedPoint();
	   aux_getPointMultiples(prec[i],pts,2*i+1);
	}
    
    // A počítáme pomocí sliding-window
    int i = exp.length-1,h,s = 0,k = 0,u;
	while (i >= 0)
	{
		if (exp.bits[i] == 0){
		  edwardsDbl(P,P);
		  i--;
		}
		else {
			s = i - w + 1;
			s = s > 0 ? s : 0;

			while (!exp.bits[s]) ++s;
			for (h = 1;h <= i-s+1;++h) square(res);

			u = buildFromNAF(coeff,s,i);

			getPrecomputed(temp,u);
			multiply(res,temp);
			counter++;
			i = s-1;
		}
	}
    
        
    
    // Zkontroluj chyby
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
      fprintf(stderr, "Launch failed: %s\n", hipGetErrorString(cudaStatus));
    
    // Synchronizovat vše
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
	  fprintf(stderr, "Launch failed: %s\n", hipGetErrorString(cudaStatus));
 
    // Zkopírovat data zpět do počítače a uvolnit paměť
    for (int i = 0;i < NUM_CURVES;++i){
      pts[i].toHost(initPoints+i);
      delete pts[i];
    }
    delete[] pts;
    
    return cudaStatus;
}

#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
using namespace std;

#include "modular.h"

__global__ void myKernel(biguint_t a, biguint_t b, biguint_t n,biguint_t c)
{
    __shared__ VOL carry_t b_cy[NB_DIGITS]; 
	__shared__ VOL digit_t r[NB_DIGITS];

	memset((void*)r,0,NB_DIGITS*sizeof(int));
	memset((void*)b_cy,0,NB_DIGITS*sizeof(int));

	Cuda_Mul_mod(c,b_cy,a,b,r,n[threadIdx.x],2047647423);
	Cuda_Add_mod(c,b_cy,b);
}

extern "C" hipError_t testCuda(biguint_t a,biguint_t b,biguint_t c,biguint_t n)
{
	void* devA = NULL,*devB = NULL,*devC = NULL,*devN = NULL;
    hipError_t cudaStatus;
    
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return cudaStatus;
    }

    hipMalloc((void**)&devA, MAX_BYTES);
    hipMalloc((void**)&devB, MAX_BYTES);
	hipMalloc((void**)&devC, MAX_BYTES);
	hipMalloc((void**)&devN, MAX_BYTES);
    
    // Copy input vectors from host memory to GPU buffers.
	hipMemcpy(devA, (void*)a, MAX_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(devB, (void*)b, MAX_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(devC, (void*)c, MAX_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(devN, (void*)n, MAX_BYTES, hipMemcpyHostToDevice);

    // Launch a kernel on the GPU with one thread for each element.
	myKernel<<<1, NB_DIGITS>>>((unsigned int*)devA, (unsigned int*)devB, (unsigned int*)devN,(unsigned int*)devC);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
      fprintf(stderr, "Launch failed: %s\n", hipGetErrorString(cudaStatus));
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
	  fprintf(stderr, "Launch failed: %s\n", hipGetErrorString(cudaStatus));

    // Copy output vector from GPU buffer to host memory.
	hipMemcpy((void*)c, devC, MAX_BYTES, hipMemcpyDeviceToHost);
    
    hipFree(devC);
    hipFree(devA);
    hipFree(devB);
    
    return cudaStatus;
}

//////////////////////////////////////////////////////////////////////////////////////

void printBigInt(biguint_t B)
{
	for (int i = 0;i < NB_DIGITS;++i)
	{
		printf("%#010x",B[i]);
		if (B[i+1] == 0) break;
		if (i != NB_DIGITS-1) printf(",");
	}
	printf("\n");
}

int main()
{
	// A,B jsou v Montgomeryho reprezentaci, A,B,N v bázi 2^32
	// N = 215714093118538583256769
	// A = 21799067859837164737
	// B = 104402829964868711809

	biguint_t A = {0x0e6c2ef9,0x9e4d4f27,0x0000299e,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000};
	biguint_t B = {0x47769205,0xdceddf18,0x00002c89,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000};
	biguint_t N = {0x1b8a2ec1,0xe2695510,0x00002dad,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000};
	biguint_t C = {0};

    hipError_t cudaStatus = testCuda(A,B,C,N);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    
	printBigInt(C);
	
	char c;
	cin >> c;
    return 0;
}

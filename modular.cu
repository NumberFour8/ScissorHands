#include "hip/hip_runtime.h"
#include "def.h"

#define __add_cc(r,a,b) ASM ("add.cc.u32 %0, %1, %2;": "=r"(r): "r"(a), "r"(b)) 
#define __addc_cc(r,a,b) ASM ("addc.cc.u32 %0, %1, %2;": "=r"(r): "r"(a), "r"(b))
#define __sub_cc(r,a,b) ASM ("sub.cc.u32 %0, %1, %2;": "=r"(r): "r"(a), "r"(b)) 

#define __addcy(carry) ASM ("addc.s32 %0, 0, 0;": "=r"(carry)) 
#define __addcy2(carry) ASM ("addc.cc.s32 %0, %0, 0;": "+r"(carry)) 

#define __subcy(carry) ASM ("subc.s32 %0, 0, 0;": "=r"(carry)) 
#define __subcy2(carry) ASM ("subc.s32 %0, %0, 0;": "+r"(carry)) 

#define __mul_lo(r,a,b) ASM("mul.lo.u32 %0, %1, %2;": "=r"(r): "r"(a),"r"(b)) 
#define __mul_hi(r,a,b) ASM("mul.hi.u32 %0, %1, %2;": "=r"(r): "r"(a),"r"(b)) 
#define __mad_lo_cc(r,a,b) ASM("mad.lo.cc.u32 %0, %1, %2, %0;":\
                                                      "+r"(r): "r"(a),"r"(b)) 
#define __madc_hi_cc(r,a,b) ASM("madc.hi.cc.u32 %0, %1, %2, %0;":\
                                                  "+r"(r):"r"(a),"r"(b)) 
     

__device__ void Cuda_Fully_Normalize (biguint_t A, bigint_t cy)
{
  carry_t cytemp;
  unsigned int thm1;

  while(__any(cy[threadIdx.x])!=0)
  {
    thm1 = (threadIdx.x - 1) % NB_DIGITS;
    cytemp = cy[thm1];

    __add_cc(A[threadIdx.x], A[threadIdx.x], cytemp);
  
    if (cytemp >= 0)
      __addcy(cy[threadIdx.x]);
    else /* if (cytemp < 0) */
      __subcy(cy[threadIdx.x]);
  }
}

/* Compute Rmod <- A + B */ 
/* Input: 0 <= A, B < 3*N */ 
/* Ouput: 0 <= Rmod < 6*N */ 
__device__ void Cuda_Add_mod
(biguint_t Rmod, bigint_t cy, const biguint_t A, const biguint_t B)
{
  unsigned int thp1 = (threadIdx.x + 1) % NB_DIGITS;
  __add_cc (Rmod[threadIdx.x], A[threadIdx.x], B[threadIdx.x]);
  __addcy2(Rmod[thp1]); 
  __addcy (cy[thp1]);
  Cuda_Fully_Normalize (Rmod, cy); 
}

/* Compute Rmod <- Rmod + B */ 
/* Input: 0 <= Rmod, B < 3*N */ 
/* (except when it follows Cuda_Mulint_mod, 0 <= Rmod < 3*N, 0 < B < 7*N ) */ 
/* Ouput: 0 <= Rmod < 6*N */ 
/* (except when it follows Cuda_Mulint_mod, 0 <= Rmod < 10*N) */ 
__device__ void Cuda_Add_mod
(biguint_t Rmod, bigint_t cy, const biguint_t A)
{
  unsigned int thp1 = (threadIdx.x + 1) % NB_DIGITS;
  __add_cc (Rmod[threadIdx.x], Rmod[threadIdx.x], A[threadIdx.x]);
  //__addcy (cy[threadIdx.x]);
  __addcy2(Rmod[thp1]); 
  __addcy (cy[thp1]);
  Cuda_Fully_Normalize (Rmod, cy);
}

/* Compute Rmod <- Rmod - B */ 
/* Input: 0 <= Rmod, B < 3*N */ 
/* Ouput: 0 <= Rmod < 6*N */ 
__device__ void Cuda_Sub_mod 
(biguint_t Rmod, bigint_t cy, const biguint_t B, const digit_t N3thdx)
{
  digit_t reg_Rmod = Rmod[threadIdx.x];
  carry_t reg_cy = 0; 
  
  __add_cc (reg_Rmod, reg_Rmod, N3thdx);
  __addcy (reg_cy);
  __sub_cc (reg_Rmod, reg_Rmod, B[threadIdx.x]);
  __subcy2 (reg_cy);

  Rmod[threadIdx.x] = reg_Rmod;
  cy[threadIdx.x] = reg_cy;
  Cuda_Fully_Normalize (Rmod, cy); 
}

/* Perform one step of REDC */ 
__device__ void Cuda_Mulmod_step
(biguint_t r, bigint_t cy, digit_t a, digit_t b, const digit_t Nthdx,
 const digit_t invN)
{
  digit_t t;
  digit_t reg_hi = 0;
  unsigned int thp1= (threadIdx.x + 1) % NB_DIGITS;
  carry_t reg_cy = cy[thp1];

  __mad_lo_cc(r[threadIdx.x],a,b);
  __madc_hi_cc(reg_hi,a,b);
  __addcy2(reg_cy);

  __mul_lo(t, invN, r[0]);
  __mad_lo_cc(r[threadIdx.x],t,Nthdx);
  __madc_hi_cc(reg_hi,t,Nthdx);
  __addcy2(reg_cy);

  /* make one round of normalize + a right shift at the same time */
  __add_cc(r[threadIdx.x],r[thp1],reg_hi);
  __addc_cc(r[thp1],r[thp1],reg_cy);
  __addcy(cy[thp1]); 
}

/* Compute r <- 2*a */ 
/* Input: 0 <= a < 3*N */ 
/* Ouput: 0 <= r < 3*N */ 
__device__ void Cuda_Dbl_mod
(biguint_t r, biguint_t a)
{
  unsigned int thp1= (threadIdx.x + 1) % NB_DIGITS;
  asm ("add.cc.u32 %0, %1, %1;" : "=r"(r[threadIdx.x]) : "r"(a[threadIdx.x]));
  __addcy2(r[thp1]);
}


/* Compute r <- A*b */ 
/* Input: 0 < b < 2^SIZE_DIGIT, 0 <= A < 6*N */ 
/* Ouput: 0 <= r < 7*N */ 
__device__ void Cuda_Mulint_mod
(biguint_t r, bigint_t cy, biguint_t A, digit_t b, const digit_t Nthdx,const digit_t invN)
{
  digit_t t;
  digit_t reg_hi;
  unsigned int thp1= (threadIdx.x + 1) % NB_DIGITS;
  digit_t reg_A = A[threadIdx.x];
  carry_t reg_cy;

  __mul_lo(r[threadIdx.x],reg_A,b);
  __mul_hi(reg_hi,reg_A,b);

  __mul_lo(t, invN, r[0]);
  __mad_lo_cc(r[threadIdx.x],t,Nthdx);
  __madc_hi_cc(reg_hi,t,Nthdx);
  __addcy(reg_cy);

  /* make one round of normalize + a right shift at the same time */
  __add_cc(r[threadIdx.x],r[thp1],reg_hi);
  __addc_cc(r[thp1],r[thp1],reg_cy);
  __addcy(cy[thp1]); 

  Cuda_Fully_Normalize(r,cy); 
}

/* Compute r <- A*B */ 
/* Input: 0 <= A, B < 6*N */
/* (except when it follows Cuda_Mulint_mod, 0 <= A < 6*N, 0 < B < 10*N ) */ 
/* Ouput: 0 <= r < 3*N */ 
__device__ void Cuda_Mul_mod 
(biguint_t mul, bigint_t cy, const biguint_t A, const biguint_t B, biguint_t r,
 const digit_t Nthdx, const digit_t invN)
{

  int i;
  digit_t temp=A[threadIdx.x];

  r[threadIdx.x]=0;
  
  for (i=0; i< NB_DIGITS; i++)
    Cuda_Mulmod_step (r, cy, temp, B[i], Nthdx, invN);

  
  Cuda_Fully_Normalize (r, cy);
  mul[threadIdx.x]=r[threadIdx.x];
}

__device__ void Cuda_Square_mod 
(biguint_t mul, bigint_t cy, const biguint_t A, biguint_t r, 
 const digit_t Nthdx, const digit_t invN)
{
  Cuda_Mul_mod (mul, cy, A, A, r, Nthdx, invN);
}

/////////////////////////////////////////////////////////////

__global__ void myKernel(biguint_t a, biguint_t b, biguint_t n,biguint_t c)
{
    __shared__ VOL carry_t b_cy[NB_DIGITS]; 
	__shared__ VOL digit_t r[NB_DIGITS];

	memset((void*)r,0,NB_DIGITS*sizeof(int));
	memset((void*)b_cy,0,NB_DIGITS*sizeof(int));

	Cuda_Mul_mod(c,b_cy,a,b,r,n[threadIdx.x],2047647423);
	Cuda_Add_mod(c,b_cy,b);
}

extern "C" hipError_t testCuda(biguint_t a,biguint_t b,biguint_t c,biguint_t n)
{
	void* devA = NULL,*devB = NULL,*devC = NULL,*devN = NULL;
    hipError_t cudaStatus;
    
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return cudaStatus;
    }

    hipMalloc((void**)&devA, MAX_BYTES);
    hipMalloc((void**)&devB, MAX_BYTES);
	hipMalloc((void**)&devC, MAX_BYTES);
	hipMalloc((void**)&devN, MAX_BYTES);
    
    // Copy input vectors from host memory to GPU buffers.
	hipMemcpy(devA, (void*)a, MAX_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(devB, (void*)b, MAX_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(devC, (void*)c, MAX_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(devN, (void*)n, MAX_BYTES, hipMemcpyHostToDevice);

    // Launch a kernel on the GPU with one thread for each element.
	myKernel<<<1, NB_DIGITS>>>((unsigned int*)devA, (unsigned int*)devB, (unsigned int*)devN,(unsigned int*)devC);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
      fprintf(stderr, "Launch failed: %s\n", hipGetErrorString(cudaStatus));
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
	  fprintf(stderr, "Launch failed: %s\n", hipGetErrorString(cudaStatus));

    // Copy output vector from GPU buffer to host memory.
	hipMemcpy((void*)c, devC, MAX_BYTES, hipMemcpyDeviceToHost);
    
    hipFree(devC);
    hipFree(devA);
    hipFree(devB);
    
    return cudaStatus;
}
